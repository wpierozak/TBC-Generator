#include "hip/hip_runtime.h"
#include"CM_postproc_GPU.cuh"

__constant__ cm_pos dimX;
__constant__ cm_pos dimY;
__constant__ cm_pos dimZ;

void fillColorBuffer_GPU(GeneratorConfig& caDomain)
{
    cm_state * domain = caDomain.getAbuffer();
    cm_size grainNum = caDomain.getNucleusNum();
    cm_colorampl* colorsArray = defineColors(grainNum);

    cm_size free, total;
    hipMemGetInfo(&free, &total);

    hipMemcpyToSymbol(HIP_SYMBOL(dimX), &dimX, sizeof(cm_pos));
    hipMemcpyToSymbol(HIP_SYMBOL(dimY), &dimY, sizeof(cm_pos));
    hipMemcpyToSymbol(HIP_SYMBOL(dimZ), &dimZ, sizeof(cm_pos));

    cm_size cellSize = caDomain.getCellSizeInBytes();
    cm_size colorsSize = caDomain.getColorsSizeInBytes();
    cm_size cellsNum = caDomain.getCellsNum();

    int chunkNum = ((cellSize + colorsSize)*cellsNum + free - 1)/free;
    cm_size chunk = (cellsNum + chunkNum - 1)/chunkNum;

    cm_state* domain_D = nullptr;
    cm_state* colorBuffer_D = nullptr;
    cm_colorampl* colorsArray_D = nullptr;
    hipMalloc(&domain_D, chunk*cellSize);
    hipMalloc(&colorBuffer_D, chunk*colorsSize);
    hipMalloc(&colorsArray_D, colorsSize*grainNum);
    hipMemcpy(colorsArray_D, colorsArray, colorsSize*grainNum, hipMemcpyHostToDevice);

    cm_size pos = 0;
    cm_colorampl* colorBuffer = (cm_colorampl*) malloc(caDomain.getColorBufferSizeInBytes());

    dim3 blockSize = {32,1,1};
    dim3 gridSize = {(chunk + blockSize.x - 1)/blockSize.x, 1, 1};
    for(cm_size i = 0; i < chunkNum; i++)
    {
        hipMemcpy(domain_D, domain + i*chunk*cellSize, chunk*cellSize, hipMemcpyHostToDevice);
        hipMemcpy(colorBuffer_D, colorBuffer + i*chunk*colorsSize, chunk*colorsSize, hipMemcpyHostToDevice);
        assignColors<<<blockSize, gridSize>>>(domain_D, colorBuffer_D, colorsArray_D);
        hipDeviceSynchronize();
        hipMemcpy(domain + i*chunk*cellSize, domain_D, chunk*cellSize, hipMemcpyDeviceToHost);
        hipMemcpy(colorBuffer + i*chunk*colorsSize, colorBuffer_D,  chunk*colorsSize, hipMemcpyDeviceToHost);
    }

    caDomain.setColorBuffer(colorBuffer);

    hipFree(domain_D);
    hipFree(colorBuffer_D);
    hipFree(colorsArray_D);
    delete[] colorsArray;
}

__global__ void assignColors(cm_state* caDomain, cm_colorampl* rgb, const cm_colorampl* __restrict__ colorArray)
{
    int idx = threadIdx.x;
    rgb[idx*3] = colorArray[caDomain[idx]*3];
    rgb[idx*3 + 1] = colorArray[caDomain[idx]*3 + 1];
    rgb[idx*3 + 2] = colorArray[caDomain[idx]*3 + 2];
}